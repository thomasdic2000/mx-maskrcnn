#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2017 by Contributors
 * \file roi_align.cu
 * \brief roi align operator
 * \author Yuchen Guo, Zehao Shi
*/
#include "./roi_align-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>

namespace mshadow {
namespace cuda {

template<typename Dtype>
__global__ void ROIAlignForwardKernel(const int count, const Dtype* bottom_data,
                                     const float spatial_scale,
                                     const int channels, const int height, const int width,
                                     const int pooled_height, const int pooled_width,
                                     const Dtype* bottom_rois, Dtype* top_data,
                                     Dtype* argmax_data) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = bottom_rois[0];

    if (roi_batch_ind < 0) {
      top_data[index] = 0;
      argmax_data[index] = 0;
      continue;
    }

    Dtype roi_start_w = (bottom_rois[1]) * spatial_scale;
    Dtype roi_start_h = (bottom_rois[2]) * spatial_scale;
    Dtype roi_end_w = (bottom_rois[3]) * spatial_scale;
    Dtype roi_end_h = (bottom_rois[4]) * spatial_scale;

    // Force malformed ROIs to be 1x1
    Dtype roi_width = max(roi_end_w - roi_start_w, static_cast<Dtype>(1));
    Dtype roi_height = max(roi_end_h - roi_start_h, static_cast<Dtype>(1));
    Dtype bin_size_h = static_cast<Dtype>(roi_height)
                       / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = static_cast<Dtype>(roi_width)
                       / static_cast<Dtype>(pooled_width);

    Dtype hstart = static_cast<Dtype>((ph) * bin_size_h);
    Dtype wstart = static_cast<Dtype>((pw) * bin_size_w);
    Dtype hend = static_cast<Dtype>((ph + 1) * bin_size_h);
    Dtype wend = static_cast<Dtype>((pw + 1) * bin_size_w);

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, static_cast<Dtype>(0)), static_cast<Dtype>(height));
    hend = min(max(hend + roi_start_h, static_cast<Dtype>(0)), static_cast<Dtype>(height));
    wstart = min(max(wstart + roi_start_w, static_cast<Dtype>(0)), static_cast<Dtype>(width));
    wend = min(max(wend + roi_start_w, static_cast<Dtype>(0)), static_cast<Dtype>(width));
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    Dtype maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    int bottom_index = 0;
    bottom_data += (roi_batch_ind * channels + c) * height * width;
    Dtype h_stride = (hend - hstart)/3.0;
    Dtype w_stride = (wend - wstart)/3.0;
    for (Dtype h = hstart+h_stride; h <= hend-h_stride+0.01; h += max(h_stride, 0.01)) {
      for (Dtype w = wstart+w_stride; w <= wend-w_stride+0.01; w += max(w_stride, 0.01)) {
        bottom_index ++;
        int hlow = min(max(static_cast<int>(floor(h)), 0), height-1);
        int hhigh = min(max(static_cast<int>(ceil(h)), 0), height-1);
        int wleft = min(max(static_cast<int>(floor(w)), 0), width-1);
        int wright = min(max(static_cast<int>(ceil(w)), 0), width-1);
        int topleft = hlow * width + wleft;
        int topright = hlow * width + wright;
        int bottomleft = hhigh * width + wleft;
        int bottomright = hhigh * width + wright;
        
        Dtype alpha = (hlow == hhigh) ? static_cast<Dtype>(0.5) : (h - hlow) / (hhigh - hlow);
        Dtype beta = (wleft == wright) ? static_cast<Dtype>(0.5) : (w - wleft) / (wright - wleft);
        Dtype value = (1 - alpha) * (1 - beta) * bottom_data[topleft] + alpha * (1 - beta) * bottom_data[bottomleft]
                            + (1 - alpha) * beta * bottom_data[topright] + alpha * beta * bottom_data[bottomright];
        
        if (value > maxval) {
          maxval = value;
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = (Dtype)maxidx;
  }
}

template<typename Dtype>
inline void ROIAlignForward(const Tensor<gpu, 4, Dtype> &out,
                           const Tensor<gpu, 4, Dtype> &data,
                           const Tensor<gpu, 2, Dtype> &bbox,
                           const Tensor<gpu, 4, Dtype> &max_idx,
                           const float spatial_scale) {
  const Dtype *bottom_data = data.dptr_;
  const Dtype *bottom_rois = bbox.dptr_;
  Dtype *top_data = out.dptr_;
  Dtype *argmax_data = max_idx.dptr_;
  const int count = out.shape_.Size();
  const int channels = data.size(1);
  const int height = data.size(2);
  const int width = data.size(3);
  const int pooled_height = out.size(2);
  const int pooled_width = out.size(3);
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridNum, (gridSize + kMaxGridNum - 1) / kMaxGridNum);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "ROIPooling Forward");
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  ROIAlignForwardKernel<Dtype><<<dimGrid, dimBlock, 0, stream>>>(
      count, bottom_data, spatial_scale, channels, height, width,
      pooled_height, pooled_width, bottom_rois, top_data, argmax_data);
}

template<typename Dtype>
__global__ void ROIAlignBackwardAccKernel(const int count, const Dtype* top_diff,
                                         const Dtype* argmax_data, const int num_rois,
                                         const float spatial_scale,
                                         const int channels, const int height, const int width,
                                         const int pooled_height, const int pooled_width,
                                         Dtype* bottom_diff, const Dtype* bottom_rois) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) coords in bottom data
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    Dtype gradient = 0;
    // Accumulate gradient over all ROIs that pooled this element
    for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
      const Dtype* offset_bottom_rois = bottom_rois + roi_n * 5;
      int roi_batch_ind = offset_bottom_rois[0];
      // Skip if ROI's batch index doesn't match n
      if (n != roi_batch_ind) {
        continue;
      }

      Dtype roi_start_w = (offset_bottom_rois[1]) * spatial_scale;
      Dtype roi_start_h = (offset_bottom_rois[2]) * spatial_scale;
      Dtype roi_end_w = (offset_bottom_rois[3]) * spatial_scale;
      Dtype roi_end_h = (offset_bottom_rois[4]) * spatial_scale;

      // Skip if ROI doesn't include (h, w)
      const bool in_roi = (w > roi_start_w - 1.0 && w < roi_end_w + 1.0 &&
                           h > roi_start_h - 1.0 && h < roi_end_h + 1.0);
      if (!in_roi) {
        continue;
      }

      int offset = (roi_n * channels + c) * pooled_height * pooled_width;
      const Dtype* offset_top_diff = top_diff + offset;
      const Dtype* offset_argmax_data = argmax_data + offset;

      // Compute feasible set of pooled units that could have pooled
      // this bottom unit

      // Force malformed ROIs to be 1x1
      Dtype roi_width = max(roi_end_w - roi_start_w, static_cast<Dtype>(1));
      Dtype roi_height = max(roi_end_h - roi_start_h, static_cast<Dtype>(1));

      Dtype bin_size_h = static_cast<Dtype>(roi_height)
                         / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = static_cast<Dtype>(roi_width)
                         / static_cast<Dtype>(pooled_width);

      for (int ph = 0; ph < pooled_height; ++ph) {
        for (int pw = 0; pw < pooled_width; ++pw) {
          Dtype hstart = static_cast<Dtype>((ph) * bin_size_h);
          Dtype wstart = static_cast<Dtype>((pw) * bin_size_w);
          Dtype hend = static_cast<Dtype>((ph + 1) * bin_size_h);
          Dtype wend = static_cast<Dtype>((pw + 1) * bin_size_w);

          hstart = min(max(hstart + roi_start_h, static_cast<Dtype>(0)), static_cast<Dtype>(height));
          hend = min(max(hend + roi_start_h, static_cast<Dtype>(0)), static_cast<Dtype>(height));
          wstart = min(max(wstart + roi_start_w, static_cast<Dtype>(0)), static_cast<Dtype>(width));
          wend = min(max(wend + roi_start_w, static_cast<Dtype>(0)), static_cast<Dtype>(width));

          bool in_bin = (w > wstart - 1.0 && w < wend + 1.0 &&
                      h > hstart - 1.0 && h < hend + 1.0);
          if (!in_bin) {
            continue;
          }

          const int pool_index = ph * pooled_width + pw;
          int bottom_index = 0;
          Dtype h_stride = (hend - hstart)/3.0;
          Dtype w_stride = (wend - wstart)/3.0;
          for (Dtype rh = hstart+h_stride; rh <= hend-h_stride+0.01; rh += max(h_stride, 0.01)) {
            for (Dtype rw = wstart+w_stride; rw <= wend-w_stride+0.01; rw += max(w_stride, 0.01)) {
              bottom_index ++;
              if (offset_argmax_data[pool_index] != bottom_index) continue;
              // compute the integer coordinates around (h, w) for bilinear interpolation
              int hlow = min(max(static_cast<int>(floor(rh)), 0), height-1);
              int hhigh = min(max(static_cast<int>(ceil(rh)), 0), height-1);
              int wleft = min(max(static_cast<int>(floor(rw)), 0), width-1);
              int wright = min(max(static_cast<int>(ceil(rw)), 0), width-1);
              if (h != hlow && h != hhigh && w != wleft && w != wright) // (w, h) is not around (rw, rh)
                  continue;
              
              Dtype alpha = (hlow == hhigh) ? static_cast<Dtype>(0.5) : (rh - hlow) / (hhigh - hlow);
              Dtype beta = (wleft == wright) ? static_cast<Dtype>(0.5) : (rw - wleft) / (wright - wleft);
              if (h == hlow && w == wleft) gradient += offset_top_diff[pool_index] * (1 - alpha) * (1 - beta);
              else if (h == hlow && w == wright) gradient += offset_top_diff[pool_index] * (1 - alpha) * beta;
              else if (h == hhigh && w == wleft) gradient += offset_top_diff[pool_index] * alpha * (1 - beta);
              else if (h == hhigh && w == wright) gradient += offset_top_diff[pool_index] * alpha * beta;
            }
          }
        }
      }
    }
    bottom_diff[index] += gradient;
  }
}

template<typename Dtype>
inline void ROIAlignBackwardAcc(const Tensor<gpu, 4, Dtype> &in_grad,
                               const Tensor<gpu, 4, Dtype> &out_grad,
                               const Tensor<gpu, 2, Dtype> &bbox,
                               const Tensor<gpu, 4, Dtype> &max_idx,
                               const float spatial_scale) {
  const Dtype *top_diff = out_grad.dptr_;
  const Dtype *bottom_rois = bbox.dptr_;
  Dtype *bottom_diff = in_grad.dptr_;
  Dtype *argmax_data = max_idx.dptr_;
  const int count = in_grad.shape_.Size();
  const int num_rois = bbox.size(0);
  const int channels = in_grad.size(1);
  const int height = in_grad.size(2);
  const int width = in_grad.size(3);
  const int pooled_height = out_grad.size(2);
  const int pooled_width = out_grad.size(3);
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridNum, (gridSize + kMaxGridNum - 1) / kMaxGridNum);
  dim3 dimBlock(kMaxThreadsPerBlock);
  CheckLaunchParam(dimGrid, dimBlock, "ROIPooling Backward");
  hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);
  ROIAlignBackwardAccKernel<Dtype><<<dimGrid, dimBlock, 0, stream>>>(
      count, top_diff, argmax_data, num_rois, spatial_scale, channels, height, width,
      pooled_height, pooled_width, bottom_diff, bottom_rois);
}

}  // namespace cuda

template<typename Dtype>
inline void ROIAlignForward(const Tensor<gpu, 4, Dtype> &out,
                           const Tensor<gpu, 4, Dtype> &data,
                           const Tensor<gpu, 2, Dtype> &bbox,
                           const Tensor<gpu, 4, Dtype> &max_idx,
                           const float spatial_scale) {
  cuda::ROIAlignForward(out, data, bbox, max_idx, spatial_scale);
}

template<typename Dtype>
inline void ROIAlignBackwardAcc(const Tensor<gpu, 4, Dtype> &in_grad,
                               const Tensor<gpu, 4, Dtype> &out_grad,
                               const Tensor<gpu, 2, Dtype> &bbox,
                               const Tensor<gpu, 4, Dtype> &max_idx,
                               const float spatial_scale) {
  cuda::ROIAlignBackwardAcc(in_grad, out_grad, bbox, max_idx, spatial_scale);
}

}  // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(ROIAlignParam param, int dtype) {
  Operator* op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new ROIAlignOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
